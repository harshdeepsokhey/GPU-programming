#include "hip/hip_runtime.h"
/* Example 1: Add 2 vectors*/

#include "cuda_test.cuh"

__global__
void cudaAddVectorKernel(const float *a, const float *b, float *c, const uint size){
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    while(idx < size){
        c[idx] = a[idx]+ b[idx]
        idx += blockDim.x * gridDim.x;
    }
}

void cudaCallAddVectorKernel( const uint block_count, const uint per_block_thread_count, const float *a, const float *b, float *c, const uint size){
    cudaAddVectorKernel<<<block_count, per_block_thread_count>>>(a,b,c,size);
}

